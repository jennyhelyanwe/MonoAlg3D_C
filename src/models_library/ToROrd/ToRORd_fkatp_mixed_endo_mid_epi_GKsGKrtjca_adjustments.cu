#include "hip/hip_runtime.h"
#include "ToRORd_fkatp_mixed_endo_mid_epi_GKsGKrtjca_adjustments.h"
#include <stddef.h>
#include <stdint.h>

__global__ void kernel_set_model_initial_conditions(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    // Default initial conditions (endocardium cell)
    if (threadID < num_volumes) {
        for (int i = 0; i < NEQ; i++) {
            // Steady-state after 200 beats (endocardium cell)
            *((real * )((char *) sv + pitch * 0) + threadID) = -8.890585e+01;
            *((real * )((char *) sv + pitch * 1) + threadID) = 1.107642e-02;
            *((real * )((char *) sv + pitch * 2) + threadID) = 6.504164e-05;
            *((real * )((char *) sv + pitch * 3) + threadID) = 1.210818e+01;
            *((real * )((char *) sv + pitch * 4) + threadID) = 1.210851e+01;
            *((real * )((char *) sv + pitch * 5) + threadID) = 1.426206e+02;
            *((real * )((char *) sv + pitch * 6) + threadID) = 1.426205e+02;
            *((real * )((char *) sv + pitch * 7) + threadID) = 1.530373e+00;
            *((real * )((char *) sv + pitch * 8) + threadID) = 1.528032e+00;
            *((real * )((char *) sv + pitch * 9) + threadID) = 7.455488e-05;
            *((real * )((char *) sv + pitch * 10) + threadID) = 7.814592e-04;
            *((real * )((char *) sv + pitch * 11) + threadID) = 8.313839e-01;
            *((real * )((char *) sv + pitch * 12) + threadID) = 8.311938e-01;
            *((real * )((char *) sv + pitch * 13) + threadID) = 6.752873e-01;
            *((real * )((char *) sv + pitch * 14) + threadID) = 8.308255e-01;
            *((real * )((char *) sv + pitch * 15) + threadID) = 1.585610e-04;
            *((real * )((char *) sv + pitch * 16) + threadID) = 5.294475e-01;
            *((real * )((char *) sv + pitch * 17) + threadID) = 2.896996e-01;
            *((real * )((char *) sv + pitch * 18) + threadID) = 9.419166e-04;
            *((real * )((char *) sv + pitch * 19) + threadID) = 9.996194e-01;
            *((real * )((char *) sv + pitch * 20) + threadID) = 5.938602e-01;
            *((real * )((char *) sv + pitch * 21) + threadID) = 4.799180e-04;
            *((real * )((char *) sv + pitch * 22) + threadID) = 9.996194e-01;
            *((real * )((char *) sv + pitch * 23) + threadID) = 6.543754e-01;
            *((real * )((char *) sv + pitch * 24) + threadID) = -2.898677e-33;
            *((real * )((char *) sv + pitch * 25) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 26) + threadID) = 9.389659e-01;
            *((real * )((char *) sv + pitch * 27) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 28) + threadID) = 9.999003e-01;
            *((real * )((char *) sv + pitch * 29) + threadID) = 9.999773e-01;
            *((real * )((char *) sv + pitch * 30) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 31) + threadID) = 1.000000e+00;
            *((real * )((char *) sv + pitch * 32) + threadID) = 4.920606e-04;
            *((real * )((char *) sv + pitch * 33) + threadID) = 8.337021e-04;
            *((real * )((char *) sv + pitch * 34) + threadID) = 6.962775e-04;
            *((real * )((char *) sv + pitch * 35) + threadID) = 8.425453e-04;
            *((real * )((char *) sv + pitch * 36) + threadID) = 9.980807e-01;
            *((real * )((char *) sv + pitch * 37) + threadID) = 1.289824e-05;
            *((real * )((char *) sv + pitch * 38) + threadID) = 3.675442e-04;
            *((real * )((char *) sv + pitch * 39) + threadID) = 2.471690e-01;
            *((real * )((char *) sv + pitch * 40) + threadID) = 1.742987e-04;
            *((real * )((char *) sv + pitch * 41) + threadID) = 5.421027e-24;
            *((real * )((char *) sv + pitch * 42) + threadID) = 6.407933e-23;
        }
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * 43) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 44) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 45) + threadID) = 0.0;    // previous dt
        }
    }
}

__global__ void kernel_set_model_initial_conditions_endo_mid_epi(real *sv, int num_volumes, size_t pitch, bool use_adpt_dt, real min_dt,\
                                                real *initial_endo, real *initial_epi, real *initial_mid, real *transmurality, real *sf_Iks) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {
        for (int i = 0; i < NEQ; i++) {
            if (transmurality[threadID] == ENDO)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_endo[i];
            else if (transmurality[threadID] == EPI)
                *((real * )((char *) sv + pitch * i) + threadID) = initial_epi[i];
            else
                *((real * )((char *) sv + pitch * i) + threadID) = initial_mid[i];
        }
            
        if(use_adpt_dt) {
            *((real *)((char *)sv + pitch * 43) + threadID) = min_dt; // dt
            *((real *)((char *)sv + pitch * 44) + threadID) = 0.0;    // time_new
            *((real *)((char *)sv + pitch * 45) + threadID) = 0.0;    // previous dt
        }
    }
}

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    size_t pitch_h;

    uint8_t use_adpt_dt = (uint8_t)solver->adaptive;

    log_info("Using GPU model implemented in %s\n", __FILE__);

    uint32_t num_volumes = solver->original_num_cells;

    if(use_adpt_dt) {
        log_info("Using Adaptive timestep to solve the ODEs\n");
    } else {
        log_info("Using Fixed timestep to solve the ODEs\n");
    }

    // execution configuration
    const int GRID = (num_volumes + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t size = num_volumes * sizeof(real);

    if(use_adpt_dt)
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ + 3));
    else
        check_cuda_error(hipMallocPitch((void **)&(solver->sv), &pitch_h, size, (size_t)NEQ));

    // Get initial condition from extra_data
    real *initial_conditions_endo = NULL;
    real *initial_conditions_epi = NULL;
    real *initial_conditions_mid = NULL;
    real *transmurality = NULL;
	real *sf_Iks = NULL;
    real *initial_conditions_endo_device = NULL;
    real *initial_conditions_epi_device = NULL;
    real *initial_conditions_mid_device = NULL;
    real *transmurality_device = NULL;
	real *sf_Iks_device = NULL;

    if(solver->ode_extra_data) {
        struct extra_data_for_torord_gksgkrtjca_twave *extra_data = (struct extra_data_for_torord_gksgkrtjca_twave*)solver->ode_extra_data;
        initial_conditions_endo = extra_data->initial_ss_endo;
        initial_conditions_epi = extra_data->initial_ss_epi;
        initial_conditions_mid = extra_data->initial_ss_mid;
        transmurality = extra_data->transmurality;
		sf_Iks = extra_data->sf_IKs;
        check_cuda_error(hipMalloc((void **)&initial_conditions_endo_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_endo_device, initial_conditions_endo, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_epi_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_epi_device, initial_conditions_epi, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&initial_conditions_mid_device, sizeof(real)*NEQ));
        check_cuda_error(hipMemcpy(initial_conditions_mid_device, initial_conditions_mid, sizeof(real)*NEQ, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&sf_Iks_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(sf_Iks_device, sf_Iks, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
    }
    else {
        log_info("[INFO] You should supply a mask function to tag the cells when using this mixed model!\n");
        log_info("[INFO] Considering all cells ENDO!\n");
    }

    if (solver->ode_extra_data) {
        kernel_set_model_initial_conditions_endo_mid_epi<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt,\
                                                            initial_conditions_endo_device, initial_conditions_epi_device, initial_conditions_mid_device,\
                                                            transmurality_device, sf_Iks_device);
    }
    else {
        kernel_set_model_initial_conditions<<<GRID, BLOCK_SIZE>>>(solver->sv, num_volumes, pitch_h, use_adpt_dt, solver->min_dt);
    }
    

    check_cuda_error(hipPeekAtLastError());
    hipDeviceSynchronize();

    check_cuda_error(hipFree(initial_conditions_endo_device));
    check_cuda_error(hipFree(initial_conditions_epi_device));
    check_cuda_error(hipFree(initial_conditions_mid_device));
    check_cuda_error(hipFree(transmurality_device));

    return pitch_h;
}

extern "C" SOLVE_MODEL_ODES(solve_model_odes_gpu) {

    size_t num_cells_to_solve = ode_solver->num_cells_to_solve;
    uint32_t * cells_to_solve = ode_solver->cells_to_solve;
    real *sv = ode_solver->sv;
    real dt = ode_solver->min_dt;
    uint32_t num_steps = ode_solver->num_steps;
    bool has_extra_params = (ode_solver->ode_extra_data) ? true : false;

    // execution configuration
    const int GRID = ((int)num_cells_to_solve + BLOCK_SIZE - 1) / BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real) * num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t) * num_cells_to_solve;

    real *stims_currents_device = NULL;
    check_cuda_error(hipMalloc((void **)&stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    // the array cells to solve is passed when we are using and adaptive mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **)&cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    uint32_t num_volumes = ode_solver->original_num_cells;
    real *transmurality = NULL;
    real *transmurality_device = NULL;
	real *sf_Iks = NULL;
	real *sf_Iks_device = NULL;
    int num_extra_parameters = 20;
    real extra_par[num_extra_parameters];
    real *extra_par_device = NULL;
    
    // Get the extra data array if exists
    // Transmurality and sf_IKs mapping defined on 'extra_data' function
    if(ode_solver->ode_extra_data) {
        struct extra_data_for_torord_gksgkrtjca_twave *extra_data = (struct extra_data_for_torord_gksgkrtjca_twave*)ode_solver->ode_extra_data;
        extra_par[0]  = extra_data->INa_Multiplier; 
        extra_par[1]  = extra_data->INaL_Multiplier;
        extra_par[2]  = extra_data->INaCa_Multiplier;
        extra_par[3]  = extra_data->INaK_Multiplier;
        extra_par[4]  = extra_data->INab_Multiplier; 
        extra_par[5]  = extra_data->Ito_Multiplier;
        extra_par[6]  = extra_data->IKr_Multiplier; 
        extra_par[7]  = extra_data->IKs_Multiplier; 
        extra_par[8]  = extra_data->IK1_Multiplier;
        extra_par[9]  = extra_data->IKb_Multiplier;
        extra_par[10]  = extra_data->IKCa_Multiplier;
        extra_par[11] = extra_data->ICaL_Multiplier;  
        extra_par[12] = extra_data->ICab_Multiplier;  
        extra_par[13] = extra_data->IpCa_Multiplier;
        extra_par[14] = extra_data->ICaCl_Multiplier; 
        extra_par[15] = extra_data->IClb_Multiplier;
        extra_par[16] = extra_data->Jrel_Multiplier;
        extra_par[17] = extra_data->Jup_Multiplier;
        extra_par[18] = extra_data->aCaMK_Multiplier;
        extra_par[19] = extra_data->taurelp_Multiplier;
        sf_Iks = extra_data->sf_IKs;
        transmurality = extra_data->transmurality;

        check_cuda_error(hipMalloc((void **)&transmurality_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(transmurality_device, transmurality, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&sf_Iks_device, sizeof(real)*num_volumes));
        check_cuda_error(hipMemcpy(sf_Iks_device, sf_Iks, sizeof(real)*num_volumes, hipMemcpyHostToDevice));
        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }
    // No [extra_data] section, we consider all cells ENDO!
    else {

        // Default: initialize all current modifiers
        for (uint32_t i = 0; i < num_extra_parameters; i++) {
            if (i == 9)
                extra_par[i] = 0.0;
            else 
                extra_par[i] = 1.0;
        }
        check_cuda_error(hipMalloc((void **)&extra_par_device, sizeof(real)*num_extra_parameters));
        check_cuda_error(hipMemcpy(extra_par_device, extra_par, sizeof(real)*num_extra_parameters, hipMemcpyHostToDevice));
    }

    // Call the kernel function to solve the cellular model on the GPU
    solve_endo_mid_epi_gpu<<<GRID, BLOCK_SIZE>>>(current_t, dt, sv, stims_currents_device, cells_to_solve_device, transmurality_device, sf_Iks_device, extra_par_device,\
                                    num_cells_to_solve, num_steps, ode_solver->pitch, ode_solver->adaptive, ode_solver->abs_tol, ode_solver->rel_tol, ode_solver->max_dt, has_extra_params);

    check_cuda_error(hipPeekAtLastError());

    if (stims_currents_device) check_cuda_error(hipFree(stims_currents_device));
    if (cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if (transmurality_device) check_cuda_error(hipFree(transmurality_device));
	if (sf_Iks_device) check_cuda_error(hipFree(sf_Iks_device));
    if (extra_par_device) check_cuda_error(hipFree(extra_par_device));
}

__global__ void solve_endo_mid_epi_gpu(real cur_time, real dt, real *sv, real *stim_currents, uint32_t *cells_to_solve, real *transmurality, real *sf_Iks, real *extra_params,\
                          uint32_t num_cells_to_solve, int num_steps, size_t pitch, bool use_adpt, real abstol, real reltol, real max_dt, bool has_extra_params) {
    const real TOLERANCE = 1e-8;
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        if(!use_adpt) {
            real rDY[NEQ];
            real a[NEQ], b[NEQ];

            for(int n = 0; n < num_steps; ++n) {

                // [!] Transmurality and sf_IKs are given by the columns of ALG mesh file 
                if (has_extra_params) {
                    RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], transmurality[threadID], sf_Iks[threadID], extra_params, sv_id, dt, pitch, false);
                }
                // Default: all cells are ENDO and sf_IKs equals to 1
                else {
                    RHS_RL_gpu(a, b, sv, rDY, stim_currents[threadID], 0.0, 1.0, extra_params, sv_id, dt, pitch, false);
                }                

                // Solve variables based on its type:
                //  Non-linear = Euler
                //  Hodkin-Huxley = Rush-Larsen || Euler (if 'a' coefficient is too small)
                SOLVE_EQUATION_EULER_GPU(0);        // v        
                SOLVE_EQUATION_EULER_GPU(1);        // CaMKt    
                SOLVE_EQUATION_EULER_GPU(2);        // cass 
                SOLVE_EQUATION_EULER_GPU(3);        // nai  
                SOLVE_EQUATION_EULER_GPU(4);        // nass 
                SOLVE_EQUATION_EULER_GPU(5);        // ki   
                SOLVE_EQUATION_EULER_GPU(6);        // kss  
                SOLVE_EQUATION_EULER_GPU(7);        // cansr
                SOLVE_EQUATION_EULER_GPU(8);        // cajsr
                SOLVE_EQUATION_EULER_GPU(9);        // cai
                SOLVE_EQUATION_RUSH_LARSEN_GPU(10); // m
                SOLVE_EQUATION_RUSH_LARSEN_GPU(11); // h
                SOLVE_EQUATION_RUSH_LARSEN_GPU(12); // j
                SOLVE_EQUATION_RUSH_LARSEN_GPU(13); // hp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(14); // jp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(15); // mL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(16); // hL
                SOLVE_EQUATION_RUSH_LARSEN_GPU(17); // hLp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(18); // a
                SOLVE_EQUATION_RUSH_LARSEN_GPU(19); // iF
                SOLVE_EQUATION_RUSH_LARSEN_GPU(20); // iS
                SOLVE_EQUATION_RUSH_LARSEN_GPU(21); // ap
                SOLVE_EQUATION_RUSH_LARSEN_GPU(22); // iFp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(23); // iSp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(24); // d
                SOLVE_EQUATION_RUSH_LARSEN_GPU(25); // ff
                SOLVE_EQUATION_RUSH_LARSEN_GPU(26); // fs
                SOLVE_EQUATION_RUSH_LARSEN_GPU(27); // fcaf
                SOLVE_EQUATION_RUSH_LARSEN_GPU(28); // fcas
                SOLVE_EQUATION_RUSH_LARSEN_GPU(29); // jca
                SOLVE_EQUATION_RUSH_LARSEN_GPU(30); // ffp
                SOLVE_EQUATION_RUSH_LARSEN_GPU(31); // fcafp
                SOLVE_EQUATION_EULER_GPU(32);       // nca
                SOLVE_EQUATION_EULER_GPU(33);       // nca_i
                SOLVE_EQUATION_EULER_GPU(34);       // ikr_c0
                SOLVE_EQUATION_EULER_GPU(35);       // ikr_c1
                SOLVE_EQUATION_EULER_GPU(36);       // ikr_c2
                SOLVE_EQUATION_EULER_GPU(37);       // ikr_i
                SOLVE_EQUATION_EULER_GPU(38);       // ikr_o
                SOLVE_EQUATION_RUSH_LARSEN_GPU(39); // xs1
                SOLVE_EQUATION_RUSH_LARSEN_GPU(40); // xs2
                SOLVE_EQUATION_RUSH_LARSEN_GPU(41); // Jrel_np
                SOLVE_EQUATION_RUSH_LARSEN_GPU(42); // Jrel_p
            }
        } else {
            solve_forward_euler_gpu_adpt(sv, stim_currents[threadID], transmurality[threadID], sf_Iks[threadID], extra_params, cur_time + max_dt, sv_id, pitch, abstol, reltol, dt, max_dt);
        }
    }
}

inline __device__ void solve_forward_euler_gpu_adpt(real *sv, real stim_curr, real transmurality, real sf_Iks, real *extra_params, real final_time, int thread_id, size_t pitch, real abstol, real reltol, real min_dt, real max_dt) {

    #define DT *((real *)((char *)sv + pitch * (NEQ)) + thread_id)
    #define TIME_NEW *((real *)((char *)sv + pitch * (NEQ+1)) + thread_id)
    #define PREVIOUS_DT *((real *)((char *)sv + pitch * (NEQ+2)) + thread_id)

    real rDY[NEQ];

    real _tolerances_[NEQ];
    real _aux_tol = 0.0;
    real dt = DT;
    real time_new = TIME_NEW;
    real previous_dt = PREVIOUS_DT;

    real edos_old_aux_[NEQ];
    real edos_new_euler_[NEQ];
    real _k1__[NEQ];
    real _k2__[NEQ];
    real _k_aux__[NEQ];
    real sv_local[NEQ];

    const real _beta_safety_ = 0.8;

    const real __tiny_ = pow(abstol, 2.0);

    if(time_new + dt > final_time) {
        dt = final_time - time_new;
    }

    for(int i = 0; i < NEQ; i++) {
        sv_local[i] = *((real *)((char *)sv + pitch * i) + thread_id);
    }

    RHS_gpu(sv_local, rDY, stim_curr, transmurality, sf_Iks, extra_params, thread_id, dt, pitch, true);
    time_new += dt;

    for(int i = 0; i < NEQ; i++) {
        _k1__[i] = rDY[i];
    }

	while(1) {

		for(int i = 0; i < NEQ; i++) {
			// stores the old variables in a vector
			edos_old_aux_[i] = sv_local[i];
			// computes euler method
			edos_new_euler_[i] = _k1__[i] * dt + edos_old_aux_[i];
			// steps ahead to compute the rk2 method
			sv_local[i] = edos_new_euler_[i];
		}

		time_new += dt;

		RHS_gpu(sv_local, rDY, stim_curr, transmurality, sf_Iks, extra_params, thread_id, dt, pitch, true);
		time_new -= dt; // step back

		real greatestError = 0.0, auxError = 0.0;
		
		for(int i = 0; i < NEQ; i++) {

			// stores the new evaluation
			_k2__[i] = rDY[i];
			_aux_tol = fabs(edos_new_euler_[i]) * reltol;
			_tolerances_[i] = (abstol > _aux_tol) ? abstol : _aux_tol;

			// finds the greatest error between  the steps
			auxError = fabs(((dt / 2.0) * (_k1__[i] - _k2__[i])) / _tolerances_[i]);

			greatestError = (auxError > greatestError) ? auxError : greatestError;
		}

		/// adapt the time step
		greatestError += __tiny_;
		previous_dt = dt;

		/// adapt the time step
		dt = _beta_safety_ * dt * sqrt(1.0f / greatestError);

		if(dt < min_dt) {
			dt = min_dt;
		}
		else if(dt > max_dt) {
			dt = max_dt;
		}

		if(time_new + dt > final_time) {
			dt = final_time - time_new;
		}

		// it doesn't accept the solution or accept and risk a NaN
		if(greatestError >= 1.0f && dt > min_dt) {
			// restore the old values to do it again
			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_old_aux_[i];
			}
		
		} else {
			for(int i = 0; i < NEQ; i++) {
				_k_aux__[i] = _k2__[i];
				_k2__[i] = _k1__[i];
				_k1__[i] = _k_aux__[i];
			}

			for(int i = 0; i < NEQ; i++) {
				sv_local[i] = edos_new_euler_[i];
			}

			if(time_new + previous_dt >= final_time) {
				if(final_time == time_new) {
					break;
				} else if(time_new < final_time) {
					dt = previous_dt = final_time - time_new;
					time_new += previous_dt;
					break;
				} 	
			} else {
				time_new += previous_dt;
			}
		}
	}

    for(int i = 0; i < NEQ; i++) {
        *((real *)((char *)sv + pitch * i) + thread_id) = sv_local[i];
    }

    DT = dt;
    TIME_NEW = time_new;
    PREVIOUS_DT = previous_dt;
}

inline __device__ void RHS_gpu(real *sv, real *rDY_, real stim_current, real transmurality, real sf_Iks, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier = extra_params[0];   
    real INaL_Multiplier = extra_params[1];  
    real INaCa_Multiplier = extra_params[2];  
    real INaK_Multiplier = extra_params[3];  
    real INab_Multiplier = extra_params[4];   
    real Ito_Multiplier = extra_params[5];  
    real IKr_Multiplier = extra_params[6];   
    real IKs_Multiplier = extra_params[7];   
    real IK1_Multiplier = extra_params[8];  
    real IKb_Multiplier = extra_params[9];  
    real IKCa_Multiplier = extra_params[10];  
    real ICaL_Multiplier = extra_params[11];   
    real ICab_Multiplier = extra_params[12];   
    real IpCa_Multiplier = extra_params[13]; 
    real ICaCl_Multiplier = extra_params[14];  
    real IClb_Multiplier = extra_params[15]; 
    real Jrel_Multiplier = extra_params[16]; 
    real Jup_Multiplier = extra_params[17]; 
    real aCaMK_Multiplier = extra_params[18]; 
    real taurelp_Multiplier = extra_params[19];

    // Get the celltype for the current cell
    real celltype = transmurality;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real v;
    real CaMKt;
    real cass;
    real nai;
    real nass;
    real ki;
    real kss;
    real cansr;
    real cajsr;
    real cai;
    real m;
    real h;
    real j;
    real hp;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real ffp;
    real fcafp;
    real nca;
    real nca_i;
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_i;
    real ikr_o;
    real xs1;
    real xs2;
    real Jrel_np;
    real Jrel_p;

    if (use_adpt_dt) {
        v = sv[0];
        CaMKt = sv[1];
        cass = sv[2];
        nai = sv[3];
        nass = sv[4];
        ki = sv[5];
        kss = sv[6];
        cansr = sv[7];
        cajsr = sv[8];
        cai = sv[9];
        m = sv[10];
        h = sv[11];
        j = sv[12];
        hp = sv[13];
        jp = sv[14];
        mL = sv[15];
        hL = sv[16];
        hLp = sv[17];
        a = sv[18];
        iF = sv[19];
        iS = sv[20];
        ap = sv[21];
        iFp = sv[22];
        iSp = sv[23];
        d = sv[24];
        ff = sv[25];
        fs = sv[26];
        fcaf = sv[27];
        fcas = sv[28];
        jca = sv[29];
        ffp = sv[30];
        fcafp = sv[31];
        nca = sv[32];
        nca_i = sv[33];
        ikr_c0 = sv[34];
        ikr_c1 = sv[35];
        ikr_c2 = sv[36];
        ikr_i = sv[37];
        ikr_o = sv[38];
        xs1 = sv[39];
        xs2 = sv[40];
        Jrel_np = sv[41];
        Jrel_p = sv[42];
    } else {
        v = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt = *((real *)((char *)sv + pitch * 1) + threadID_);
        cass = *((real *)((char *)sv + pitch * 2) + threadID_);
        nai = *((real *)((char *)sv + pitch * 3) + threadID_);
        nass = *((real *)((char *)sv + pitch * 4) + threadID_);
        ki = *((real *)((char *)sv + pitch * 5) + threadID_);
        kss = *((real *)((char *)sv + pitch * 6) + threadID_);
        cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
        cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
        cai = *((real *)((char *)sv + pitch * 9) + threadID_);
        m = *((real *)((char *)sv + pitch * 10) + threadID_);
        h = *((real *)((char *)sv + pitch * 11) + threadID_);
        j = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp = *((real *)((char *)sv + pitch * 17) + threadID_);
        a = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp = *((real *)((char *)sv + pitch * 23) + threadID_);
        d = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i = *((real *)((char *)sv + pitch * 33) + threadID_);
        ikr_c0 = *((real *)((char *)sv + pitch * 34) + threadID_);
        ikr_c1 = *((real *)((char *)sv + pitch * 35) + threadID_);
        ikr_c2 = *((real *)((char *)sv + pitch * 36) + threadID_);
        ikr_i = *((real *)((char *)sv + pitch * 37) + threadID_);
        ikr_o = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1 = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2 = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);
    }

    #include "ToRORd_fkatp_mixed_endo_mid_epi_GKsGKrtjca_adjustments.common.c"
}


inline __device__ void RHS_RL_gpu(real *a_, real *b_, real *sv, real *rDY_, real stim_current, real transmurality, real sf_Iks, real *extra_params, int threadID_, real dt, size_t pitch, bool use_adpt_dt) {
    
    // Current modifiers
    real INa_Multiplier = extra_params[0];   
    real INaL_Multiplier = extra_params[1];  
    real INaCa_Multiplier = extra_params[2];  
    real INaK_Multiplier = extra_params[3];  
    real INab_Multiplier = extra_params[4];   
    real Ito_Multiplier = extra_params[5];  
    real IKr_Multiplier = extra_params[6];   
    real IKs_Multiplier = extra_params[7];   
    real IK1_Multiplier = extra_params[8];  
    real IKb_Multiplier = extra_params[9];  
    real IKCa_Multiplier = extra_params[10];  
    real ICaL_Multiplier = extra_params[11];   
    real ICab_Multiplier = extra_params[12];   
    real IpCa_Multiplier = extra_params[13]; 
    real ICaCl_Multiplier = extra_params[14];  
    real IClb_Multiplier = extra_params[15]; 
    real Jrel_Multiplier = extra_params[16]; 
    real Jup_Multiplier = extra_params[17]; 
    real aCaMK_Multiplier = extra_params[18]; 
    real taurelp_Multiplier = extra_params[19];

    // Get the celltype for the current cell
    real celltype = transmurality;
    
    // Get the stimulus current from the current cell
    real calc_I_stim = stim_current;

    // State variables
    real v;
    real CaMKt;
    real cass;
    real nai;
    real nass;
    real ki;
    real kss;
    real cansr;
    real cajsr;
    real cai;
    real m;
    real h;
    real j;
    real hp;
    real jp;
    real mL;
    real hL;
    real hLp;
    real a;
    real iF;
    real iS;
    real ap;
    real iFp;
    real iSp;
    real d;
    real ff;
    real fs;
    real fcaf;
    real fcas;
    real jca;
    real ffp;
    real fcafp;
    real nca;
    real nca_i;
    real ikr_c0;
    real ikr_c1;
    real ikr_c2;
    real ikr_i;
    real ikr_o;
    real xs1;
    real xs2;
    real Jrel_np;
    real Jrel_p;

    if (use_adpt_dt) {
        v = sv[0];
        CaMKt = sv[1];
        cass = sv[2];
        nai = sv[3];
        nass = sv[4];
        ki = sv[5];
        kss = sv[6];
        cansr = sv[7];
        cajsr = sv[8];
        cai = sv[9];
        m = sv[10];
        h = sv[11];
        j = sv[12];
        hp = sv[13];
        jp = sv[14];
        mL = sv[15];
        hL = sv[16];
        hLp = sv[17];
        a = sv[18];
        iF = sv[19];
        iS = sv[20];
        ap = sv[21];
        iFp = sv[22];
        iSp = sv[23];
        d = sv[24];
        ff = sv[25];
        fs = sv[26];
        fcaf = sv[27];
        fcas = sv[28];
        jca = sv[29];
        ffp = sv[30];
        fcafp = sv[31];
        nca = sv[32];
        nca_i = sv[33];
        ikr_c0 = sv[34];
        ikr_c1 = sv[35];
        ikr_c2 = sv[36];
        ikr_i = sv[37];
        ikr_o = sv[38];
        xs1 = sv[39];
        xs2 = sv[40];
        Jrel_np = sv[41];
        Jrel_p = sv[42];
    } else {
        v = *((real *)((char *)sv + pitch * 0) + threadID_);
        CaMKt = *((real *)((char *)sv + pitch * 1) + threadID_);
        cass = *((real *)((char *)sv + pitch * 2) + threadID_);
        nai = *((real *)((char *)sv + pitch * 3) + threadID_);
        nass = *((real *)((char *)sv + pitch * 4) + threadID_);
        ki = *((real *)((char *)sv + pitch * 5) + threadID_);
        kss = *((real *)((char *)sv + pitch * 6) + threadID_);
        cansr = *((real *)((char *)sv + pitch * 7) + threadID_);
        cajsr = *((real *)((char *)sv + pitch * 8) + threadID_);
        cai = *((real *)((char *)sv + pitch * 9) + threadID_);
        m = *((real *)((char *)sv + pitch * 10) + threadID_);
        h = *((real *)((char *)sv + pitch * 11) + threadID_);
        j = *((real *)((char *)sv + pitch * 12) + threadID_);
        hp = *((real *)((char *)sv + pitch * 13) + threadID_);
        jp = *((real *)((char *)sv + pitch * 14) + threadID_);
        mL = *((real *)((char *)sv + pitch * 15) + threadID_);
        hL = *((real *)((char *)sv + pitch * 16) + threadID_);
        hLp = *((real *)((char *)sv + pitch * 17) + threadID_);
        a = *((real *)((char *)sv + pitch * 18) + threadID_);
        iF = *((real *)((char *)sv + pitch * 19) + threadID_);
        iS = *((real *)((char *)sv + pitch * 20) + threadID_);
        ap = *((real *)((char *)sv + pitch * 21) + threadID_);
        iFp = *((real *)((char *)sv + pitch * 22) + threadID_);
        iSp = *((real *)((char *)sv + pitch * 23) + threadID_);
        d = *((real *)((char *)sv + pitch * 24) + threadID_);
        ff = *((real *)((char *)sv + pitch * 25) + threadID_);
        fs = *((real *)((char *)sv + pitch * 26) + threadID_);
        fcaf = *((real *)((char *)sv + pitch * 27) + threadID_);
        fcas = *((real *)((char *)sv + pitch * 28) + threadID_);
        jca = *((real *)((char *)sv + pitch * 29) + threadID_);
        ffp = *((real *)((char *)sv + pitch * 30) + threadID_);
        fcafp = *((real *)((char *)sv + pitch * 31) + threadID_);
        nca = *((real *)((char *)sv + pitch * 32) + threadID_);
        nca_i = *((real *)((char *)sv + pitch * 33) + threadID_);
        ikr_c0 = *((real *)((char *)sv + pitch * 34) + threadID_);
        ikr_c1 = *((real *)((char *)sv + pitch * 35) + threadID_);
        ikr_c2 = *((real *)((char *)sv + pitch * 36) + threadID_);
        ikr_i = *((real *)((char *)sv + pitch * 37) + threadID_);
        ikr_o = *((real *)((char *)sv + pitch * 38) + threadID_);
        xs1 = *((real *)((char *)sv + pitch * 39) + threadID_);
        xs2 = *((real *)((char *)sv + pitch * 40) + threadID_);
        Jrel_np = *((real *)((char *)sv + pitch * 41) + threadID_);
        Jrel_p = *((real *)((char *)sv + pitch * 42) + threadID_);
    }

    #include "ToRORd_fkatp_mixed_endo_mid_epi_GKsGKrtjca_adjustments_RL.common.c"
}
